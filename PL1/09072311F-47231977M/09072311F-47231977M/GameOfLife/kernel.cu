#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include "./common/book.h"

#include <time.h>
#include <stdio.h>
#include <iostream>
#include <stdlib.h>

#define MUERTA "\x1b[34m"
#define VIVA "\x1b[36m"
#define RESET "\x1b[0m"

__global__ void movimientoCelular(char* matriz, char* matrizResultado, int fila, int columna);

hipError_t lanzarKernel(char* matriz, char* matrizResultado, int fila, int columna);

int contarVivas(char* matriz, int dimension);

void imprimirMatriz(char* matriz, int dimension, int columna);

void rellenarMatriz(char* matriz, int dimension);

int main(int arg, char* argv[])
{

    //Comprueba que haya solo el numero de argumento permitidos
    if (arg != 4) {
        printf("\nERROR: El numero de argumentos es erroneo (.exe <-a/-m> <fila> <columna>)\n");
    }
    else {

        //Conversion de argumentos a int
        char* filaPuntero = argv[2];
        int fila = atoi(filaPuntero);
        char* columnaPuntero = argv[3];
        int columna = atoi(columnaPuntero);

        //Inicializamos hipDeviceProp_t para coger las propiedades de la tarjeta
        hipDeviceProp_t propiedades;
        HANDLE_ERROR(hipGetDeviceProperties(&propiedades, 0));

        //Dimension de la matriz
        int dimension = columna * fila;

        //Matrices
        char* matriz = NULL;
        char* matrizResultado = NULL;

        matriz = (char*)malloc(sizeof(char) * dimension);
        matrizResultado = (char*)malloc(sizeof(char) * dimension);

        //Booleano para saber si el usuario quiere manual o automatico, por defecto automatico
        bool manual = false;

        //Comprueba que los numeros de columna y fila son correctos
        if (columna <= 0 | fila <= 0) {
            printf("\nERROR: La fila/columna tiene que ser un entero positivo.\n");
        }
        //Comprueba que se haya introducido el parametro de ejecucion correcto 
        else if ((strcmp("-m", argv[1]) & strcmp("-a", argv[1])) != 0) {
            printf("\nERROR: Argumentos validos solo -m[manual] o -a[automatico]\n");
        }
        else if (propiedades.maxThreadsPerBlock < dimension) {
            printf("\nERROR: Numero de bloques supera el maximo permitido por su tarjeta.\n");
        }
        //Una vez comprobado todo empezamos con la ejecucion
        else {

            printf("\n[Matriz(%dx%d) Dimension(%d)] [modo: %s] \n", fila, columna, dimension, argv[1]);

            if (strcmp("-m", argv[1]) == 0) {
                manual = true;
            }

            //Rellenamos el tablero con celulas muertas y vivas
            rellenarMatriz(matriz, dimension);

            printf("\n***TABLERO INICIAL***\n");
            //imprimirMatriz(matriz, dimension, columna);

            int generaciones = 1; //Cuenta cuantas iteraciones (generaciones) han habido
            int vivas = 1;

            while (vivas != dimension && vivas != 0) {

                system("CLS");

                if (generaciones == 1) {
                    lanzarKernel(matriz, matrizResultado, fila, columna);
                }
                else {
                    lanzarKernel(matrizResultado, matrizResultado, fila, columna);
                }

                vivas = contarVivas(matrizResultado, dimension);

                printf("\nGeneracion: %d\n", generaciones);
                printf("Celulas vivas: %d\n", vivas);
                imprimirMatriz(matrizResultado, dimension, columna);

                //Si el usuario marca como manual, cada generacion tendra que pulsar alguna tecla para continuar
                if (manual) {
                    system("pause");
                }
                else {
                    Sleep(1000);
                }

                generaciones++;
            }
        }

        //Liberamos los arrays
        free(matriz);
        free(matrizResultado);

    }
}

__global__ void movimientoCelular(char* matriz, char* matrizResultado, int fila, int columna) {

    int posicion = threadIdx.x * columna + threadIdx.y;

    int contador = 0;

    //Primera fila 0x
    if (threadIdx.x == 0) {
        //Posicion esquina ariba izquierda 0x0
        if (threadIdx.y == 0) {

            if ((matriz[posicion + 1]) == 'X') { contador++; }
            if ((matriz[posicion + columna]) == 'X') { contador++; }
            if ((matriz[posicion + (columna + 1)]) == 'X') { contador++; }
        }
        //Posicion esquina superior derecha
        else if (threadIdx.y == (columna - 1)) {

            if ((matriz[posicion - 1]) == 'X') { contador++; }
            if ((matriz[posicion + columna]) == 'X') { contador++; }
            if ((matriz[posicion + (columna - 1)]) == 'X') { contador++; }
        }
        //Posicion en la primera fila sin contar esquinas
        else {

            if ((matriz[posicion - 1]) == 'X') { contador++; }
            if ((matriz[posicion + 1]) == 'X') { contador++; }
            if ((matriz[posicion + columna]) == 'X') { contador++; }
            if ((matriz[posicion + (columna - 1)]) == 'X') { contador++; }
            if ((matriz[posicion + (columna + 1)]) == 'X') { contador++; }
        }
    }
    //Ulima fila finalXx
    else if (threadIdx.x == (fila - 1)) {
        //Posicion esquina abajo izquierda
        if (threadIdx.y == 0) {

            if ((matriz[posicion + 1]) == 'X') { contador++; }
            if ((matriz[posicion - columna]) == 'X') { contador++; }
            if ((matriz[posicion - (columna - 1)]) == 'X') { contador++; }
        }
        //Posicion esquina abajo derecha
        else if (threadIdx.y == (columna - 1)) {

            if ((matriz[posicion - 1]) == 'X') { contador++; }
            if ((matriz[posicion - columna]) == 'X') { contador++; }
            if ((matriz[posicion - (columna + 1)]) == 'X') { contador++; }
        }
        //Posiciones ultima fila entre esquinas
        else {

            if ((matriz[posicion - 1]) == 'X') { contador++; }
            if ((matriz[posicion + 1]) == 'X') { contador++; }
            if ((matriz[posicion - columna]) == 'X') { contador++; }
            if ((matriz[posicion - (columna + 1)]) == 'X') { contador++; }
            if ((matriz[posicion - (columna - 1)]) == 'X') { contador++; }
        }
    }
    //Primera columna entre las dos esquinas izquierdas
    else if (threadIdx.y == 0) {

        if ((matriz[posicion + 1]) == 'X') { contador++; }
        if ((matriz[posicion - columna]) == 'X') { contador++; }
        if ((matriz[posicion + columna]) == 'X') { contador++; }
        if ((matriz[posicion + (columna + 1)]) == 'X') { contador++; }
        if ((matriz[posicion - (columna - 1)]) == 'X') { contador++; }
    }
    //Ultima colunmna xfinalY
    else if (threadIdx.y == columna - 1) {

        if ((matriz[posicion - 1]) == 'X') { contador++; }
        if ((matriz[posicion + columna]) == 'X') { contador++; }
        if ((matriz[posicion - columna]) == 'X') { contador++; }
        if ((matriz[posicion - (columna + 1)]) == 'X') { contador++; }
        if ((matriz[posicion + (columna - 1)]) == 'X') { contador++; }
    }
    //Posiciones fuera de los margenes
    else {

        if ((matriz[posicion + 1]) == 'X') { contador++; }
        if ((matriz[posicion - 1]) == 'X') { contador++; }
        if ((matriz[posicion + columna]) == 'X') { contador++; }
        if ((matriz[posicion - columna]) == 'X') { contador++; }
        if ((matriz[posicion - (columna + 1)]) == 'X') { contador++; }
        if ((matriz[posicion - (columna - 1)]) == 'X') { contador++; }
        if ((matriz[posicion + (columna + 1)]) == 'X') { contador++; }
        if ((matriz[posicion + (columna - 1)]) == 'X') { contador++; }
    }

    //VIVA
    if (matriz[posicion] == 'X') {

        if (contador == 2 || contador == 3) { matrizResultado[posicion] = 'X'; }
        else { matrizResultado[posicion] = 'O'; }
    }
    //MUERTA
    else {

        if (contador == 3) { matrizResultado[posicion] = 'X'; }
        else { matrizResultado[posicion] = 'O'; }
    }
}

hipError_t lanzarKernel(char* matriz, char* matrizResultado, int fila, int columna) {

    char* matriz_d = NULL;
    char* matrizResultado_d = NULL;

    int dimension = fila * columna;

    hipError_t cudaStatus;

    //Dimensiones del bloque
    dim3 blockDim(fila, columna);

    //Seleccionamos el device
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice fallo: Tienes una GPU instalada?");
        goto Error;
    }

    //Reservamos las memorias
    cudaStatus = hipMalloc((void**)&matriz_d, dimension * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc matriz_d fallo.");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&matrizResultado_d, dimension * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ERROR: hipMalloc matrizResultado_d fallo.");
        goto Error;
    }

    //Copiamos los vectores que entran por parametro
    cudaStatus = hipMemcpy(matriz_d, matriz, dimension * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ERROR: hipMemcpy matriz a matriz_d fallo.");
        goto Error;
    }

    cudaStatus = hipMemcpy(matrizResultado_d, matrizResultado, dimension * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ERROR: hipMemcpy matrizResultado a matrizResultado_d fallo.");
        goto Error;
    }


    //Lanzamos el kernel
    movimientoCelular << < 1, blockDim >> > (matriz_d, matrizResultado_d, fila, columna);


    //Miramos los errores al lanzar el kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ERROR: lanzamiento de kernel fallo: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    //Miramos errores despues de lanzar el kernel
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ERROR: el kernel fallo con codigo %d\n", cudaStatus);
        goto Error;
    }

    //Copiamos el resultado en nuestra matriz
    cudaStatus = hipMemcpy(matrizResultado, matrizResultado_d, dimension * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "ERROR: hipMemcpy matrizResultado_d a matrizResultado fallo.");
        goto Error;
    }


Error:
    hipFree(matriz_d);
    hipFree(matrizResultado_d);

    return cudaStatus;
}

void imprimirMatriz(char* matriz, int dimension, int columna) {

    for (int i = 0; i < dimension; i++) {

        if (matriz[i] == 'X') {
            printf(VIVA " X " RESET);
        }
        else {
            printf(MUERTA " O " RESET);
        }

        if ((i + 1) % columna == 0) {
            printf("\n");
        }
    }
}

int contarVivas(char* matriz, int dimension) {

    int contador = 0;

    for (int i = 0; i < dimension; i++) {
        if (matriz[i] == 'X') {
            contador++;
        }
    }

    return contador;
}

void rellenarMatriz(char* matriz, int dimension) {

    srand(time(0));

    for (int i = 0; i < dimension; i++) {

        char* celula = matriz + i;

        int random = rand() % dimension + 1;

        //Creacion del tablero en funcion de la dimension de este
        if (dimension <= 40) {
            if (random % 2 == 0) {

                *celula = 'X';
            }
            else {
                *celula = 'O';
            }
        }
        else if (dimension > 40) {
            if (random % 3 == 0 && random % 2 == 0) {

                *celula = 'X';
            }
            else {
                *celula = 'O';
            }
        }

    }
}